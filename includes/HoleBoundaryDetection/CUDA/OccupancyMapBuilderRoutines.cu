#include "hip/hip_runtime.h"
//
// Created by yaoyu on 5/11/20.
//

#include "HoleBoundaryDetection/CUDA/OccupancyMapBuilderRoutines.h"

const int N_THREADS_PER_BLOCK=256;

using namespace pcu;

__device__
void d_Rx_T(const CReal *xw, const CReal *R, const CReal *T, CReal *xs) {
    CReal xt[3];

    xt[0] = xw[0] - T[0];
    xt[1] = xw[1] - T[1];
    xt[2] = xw[2] - T[2];

    xs[0] = R[0] * xt[0] + R[3] * xt[1] + R[6] * xt[2];
    xs[1] = R[1] * xt[0] + R[4] * xt[1] + R[7] * xt[2];
    xs[2] = R[2] * xt[0] + R[5] * xt[1] + R[8] * xt[2];
}

__device__
void d_sp_2_pixel( const CReal* K, const CReal *xs, CReal *pixel ) {
    pixel[0] = ( K[0] * xs[0] ) / xs[2] + K[2];
    pixel[1] = ( K[4] * xs[1] ) / xs[2] + K[5];
    pixel[2] = 1.0;
}

__global__
void g_update_visibility_mask(
        CReal* pc, int nPc,
        CReal* camProj, int height, int width,
        std::uint8_t* visMask,
        CReal *pixels ) {
    const int index  = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    const int sizeMat = 9;
    const int sizeVec = 3;
    const int sizeCP  = 2*sizeMat + sizeVec;

    __shared__ CReal cp[sizeCP];

    // Thread 0 load the data into the shared memory.
    if ( 0 == threadIdx.x ) {
        for ( int i = 0; i < sizeCP; ++i ) {
            cp[i] = camProj[i];
        }
    }

    __syncthreads();

    // Re-map the variables in cp.
    CReal *camK  = cp;
    CReal *camR  = camK + sizeMat;
    CReal *camT  = camR + sizeMat;

    CReal xs[3];
    CReal xw[3];
    CReal pixel[3];

    // Check visibility
    for (int i = index; i < nPc; i+=stride) {
        xw[0] = pc[ i*3 ];
        xw[1] = pc[ i*3 + 1];
        xw[2] = pc[ i*3 + 2];

        d_Rx_T( xw, camR, camT, xs );

        if ( xs[2] <= 0 ) {
            visMask[i] = OCP_MAP_CAM_INVISIBLE; // Invisible.
            continue;
        }

        // Project to the image plane.
        d_sp_2_pixel(camK, xs, pixel);

        if ( pixel[0] < 0 || pixel[0] > width ) {
            visMask[i] = OCP_MAP_CAM_INVISIBLE; // Invisible.
            continue;
        }

        if ( pixel[1] < 0 || pixel[1] > height ) {
            visMask[i] = OCP_MAP_CAM_INVISIBLE; // Invisible.
            continue;
        }

        visMask[i] = OCP_MAP_CAM_VISIBLE; // Visible.

        pixels[ i*3 ]    = pixel[0];
        pixels[ i*3 + 1] = pixel[1];
        pixels[ i*3 + 2] = xs[2];
    }

    __syncthreads();
}

static void copy_2_device( const CReal *from, CReal *to, int n ) {
    for ( int i = 0; i < n; ++i ) {
        to[i] = from[i];
    }
}

CR_VisMask::CR_VisMask(int n)
: size(3*n), nPCPoints(n)
{
    hipMallocManaged(&uPointCloud, size * sizeof(CReal));

    const int sizeMat = 9;
    const int sizeVec = 3;
    const int sizeCamProj = sizeMat * 2 + sizeVec;
    hipMallocManaged(&uCamProj, sizeCamProj * sizeof(CReal));

    hipMallocManaged(&uVisMask, nPCPoints * sizeof(std::uint8_t));
    hipMallocManaged(&uPixels, size * sizeof(CReal));
}

CR_VisMask::~CR_VisMask() {
    hipFree(uPixels);
    hipFree(uVisMask);
    hipFree(uCamProj);
    hipFree(uPointCloud);
}

CReal* CR_VisMask::get_u_cam_proj() {
    return uCamProj;
}

void CR_VisMask::set_cam_proj_size(int h, int w) {
    height = h;
    width  = w;
}

void CR_VisMask::copy_point_cloud( const CReal *pPointCloud, int n ) {
    if ( 3*n > size ) {
        std::stringstream ss;
        ss << __FILE__ << ": "<< __LINE__ << ": Point cloud contains too many points. "
           << size << " is reserved but " << n << " is required. ";
        throw std::runtime_error(ss.str());
    } else {
        nPCPoints = n;
    }

    copy_2_device(pPointCloud, uPointCloud, 3*nPCPoints);
}

int CR_VisMask::cr_update_visibility_mask() {
    // CUDA context check.
    auto err = hipGetLastError();
    if ( hipSuccess != err )
    {
        std::stringstream ss;
        ss << __FILE__ << ": "<< __LINE__ << ": hipGetLastError() returns " << err;
        throw std::runtime_error(ss.str());
    }

    // Launch size.
    const int blocks = ( nPCPoints + N_THREADS_PER_BLOCK - 1 ) / N_THREADS_PER_BLOCK;
    g_update_visibility_mask<<<blocks, N_THREADS_PER_BLOCK>>>(
            uPointCloud, nPCPoints,
            uCamProj, height, width,
            uVisMask, uPixels );

    // Wait for the GPU.
    hipDeviceSynchronize();

    // CUDA context check.
    err = hipGetLastError();
    if ( hipSuccess != err )
    {
        std::stringstream ss;
        ss << __FILE__ << ": "<< __LINE__ << ": hipGetLastError() returns " << err;
        throw std::runtime_error(ss.str());
    }

    return 0;
}

std::uint8_t* CR_VisMask::get_vis_mask() {
    return uVisMask;
}

CReal* CR_VisMask::get_pixels() {
    return uPixels;
}