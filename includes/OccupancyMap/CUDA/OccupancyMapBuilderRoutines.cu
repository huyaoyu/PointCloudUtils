#include "hip/hip_runtime.h"
//
// Created by yaoyu on 5/11/20.
//

#include <fstream>
#include <sstream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "OccupancyMapBuilderRoutines.h"

const int N_THREADS_PER_BLOCK=256;

using namespace pcu;

__device__
void d_Rx_T(const CReal *xw, const CReal *R, const CReal *T, CReal *xs) {
    CReal xt[3];

    xt[0] = xw[0] - T[0];
    xt[1] = xw[1] - T[1];
    xt[2] = xw[2] - T[2];

    xs[0] = R[0] * xt[0] + R[3] * xt[1] + R[6] * xt[2];
    xs[1] = R[1] * xt[0] + R[4] * xt[1] + R[7] * xt[2];
    xs[2] = R[2] * xt[0] + R[5] * xt[1] + R[8] * xt[2];
}

__device__
void d_sp_2_pixel( const CReal* K, const CReal *xs, CReal *pixel ) {
    pixel[0] = ( K[0] * xs[0] ) / xs[2] + K[2];
    pixel[1] = ( K[4] * xs[1] ) / xs[2] + K[5];
    pixel[2] = 1.0;
}

__global__
void g_update_visibility_mask(
        CReal* pc, int nPc,
        CReal* camProj, int height, int width,
        CMask* visMask,
        CReal *pixels ) {
    const int index  = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    const int sizeMat = 9;
    const int sizeVec = 3;
    const int sizeCP  = 2*sizeMat + sizeVec;

    __shared__ CReal cp[sizeCP];

    // Thread 0 load the data into the shared memory.
    if ( 0 == threadIdx.x ) {
        for ( int i = 0; i < sizeCP; ++i ) {
            cp[i] = camProj[i];
        }
    }

    __syncthreads();

    // Re-map the variables in cp.
    CReal *camK  = cp;
    CReal *camR  = camK + sizeMat;
    CReal *camT  = camR + sizeMat;

    CReal xs[3];
    CReal xw[3];
    CReal pixel[3];

    // Check visibility
    for (int i = index; i < nPc; i+=stride) {
        xw[0] = pc[ i*3 ];
        xw[1] = pc[ i*3 + 1];
        xw[2] = pc[ i*3 + 2];

        d_Rx_T( xw, camR, camT, xs );

        if ( xs[2] <= 0 ) {
            visMask[i] = OCP_MAP_CAM_INVISIBLE; // Invisible.
            continue;
        }

        // Project to the image plane.
        d_sp_2_pixel(camK, xs, pixel);

        if ( pixel[0] < 0 || pixel[0] > width ) {
            visMask[i] = OCP_MAP_CAM_INVISIBLE; // Invisible.
            continue;
        }

        if ( pixel[1] < 0 || pixel[1] > height ) {
            visMask[i] = OCP_MAP_CAM_INVISIBLE; // Invisible.
            continue;
        }

        visMask[i] = OCP_MAP_CAM_VISIBLE; // Visible.

        pixels[ i*3 ]    = pixel[0];
        pixels[ i*3 + 1] = pixel[1];
        pixels[ i*3 + 2] = xs[2];
    }

    __syncthreads();
}

static void copy_2_device( const CReal *from, CReal *to, int n ) {
    for ( int i = 0; i < n; ++i ) {
        to[i] = from[i];
    }
}

CR_VisMask::CR_VisMask(int n)
: size(3*n), nPCPoints(n)
{
    hipMallocManaged(&uPointCloud, size * sizeof(CReal));

    const int sizeMat = 9;
    const int sizeVec = 3;
    const int sizeCamProj = sizeMat * 2 + sizeVec;
    hipMallocManaged(&uCamProj, sizeCamProj * sizeof(CReal));

    hipMallocManaged(&uVisMask, nPCPoints * sizeof(CMask));
    hipMallocManaged(&uPixels, size * sizeof(CReal));
}

CR_VisMask::~CR_VisMask() {
    hipFree(uPixels);
    hipFree(uVisMask);
    hipFree(uCamProj);
    hipFree(uPointCloud);
}

CReal* CR_VisMask::get_u_cam_proj() {
    return uCamProj;
}

void CR_VisMask::set_cam_proj_size(int h, int w) {
    height = h;
    width  = w;
}

void CR_VisMask::copy_point_cloud( const CReal *pPointCloud, int n ) {
    if ( 3*n > size ) {
        std::stringstream ss;
        ss << __FILE__ << ": "<< __LINE__ << ": Point cloud contains too many points. "
           << size << " is reserved but " << n << " is required. ";
        throw std::runtime_error(ss.str());
    } else {
        nPCPoints = n;
    }

    copy_2_device(pPointCloud, uPointCloud, 3*nPCPoints);
}

int CR_VisMask::cr_update_visibility_mask() {
    // CUDA context check.
    auto err = hipGetLastError();
    if ( hipSuccess != err )
    {
        std::stringstream ss;
        ss << __FILE__ << ": "<< __LINE__ << ": hipGetLastError() returns " << err;
        throw std::runtime_error(ss.str());
    }

    // Launch size.
    const int blocks = ( nPCPoints + N_THREADS_PER_BLOCK - 1 ) / N_THREADS_PER_BLOCK;
    g_update_visibility_mask<<<blocks, N_THREADS_PER_BLOCK>>>(
            uPointCloud, nPCPoints,
            uCamProj, height, width,
            uVisMask, uPixels );

    // Wait for the GPU.
    hipDeviceSynchronize();

    // CUDA context check.
    err = hipGetLastError();
    if ( hipSuccess != err )
    {
        std::stringstream ss;
        ss << __FILE__ << ": "<< __LINE__ << ": hipGetLastError() returns " << err;
        throw std::runtime_error(ss.str());
    }

    return 0;
}

CMask* CR_VisMask::get_vis_mask() {
    return uVisMask;
}

CReal* CR_VisMask::get_pixels() {
    return uPixels;
}

// ============================================================

void CR_DenseGrid::resize(int vx, int vy, int vz) {
    nx = vx;
    ny = vy;
    nz = vz;
    denseGrid.resize(static_cast<std::size_t>(nz) * ny * nx );
    thrust::fill( denseGrid.begin(), denseGrid.end(), OCP_MAP_OCC_UNKNOWN );
    std::cout << "denseGrid.size() = " << denseGrid.size() << std::endl;
}

CMask* CR_DenseGrid::get_dense_grid() {
    return thrust::raw_pointer_cast( denseGrid.data() );
}

typedef struct  {
    int x;
    int y;
    int z;
} DenseGridDim_t;

__global__
void g_find_frontiers( CMask *cmask, DenseGridDim_t dgd ) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int strideX = blockDim.x * gridDim.x;
    const int strideY = blockDim.y * gridDim.y;
    const int strideZ = blockDim.z * gridDim.z;

    const int dimXY = dgd.x * dgd.y;

    const int kDimXP  = blockDim.x + 2; // Kernel dimension x padded.
    const int kDimYP  = blockDim.y + 2;
//    const int kDimZP  = blockDim.z + 2;
    const int kDimXYP = kDimXP * kDimYP;
    const int kIdx = (threadIdx.z+1) * kDimXYP + (threadIdx.y+1) * kDimXP + (threadIdx.x+1);

    extern __shared__ CMask shared_mask_array[];

    const int blockXEnd = ( ( dgd.x + strideX - 1 ) / strideX ) * strideX;
    const int blockYEnd = ( ( dgd.y + strideY - 1 ) / strideY ) * strideY;
    const int blockZEnd = ( ( dgd.z + strideZ - 1 ) / strideZ ) * strideZ;

    const int nNeighbors = 26;
    int neighborShift[nNeighbors] = {
            /* Group -1, 9 neighbors. */
            -kDimXYP-kDimXP-1, -kDimXYP-kDimXP, -kDimXYP-kDimXP+1,
            -kDimXYP       -1, -kDimXYP       , -kDimXYP       +1,
            -kDimXYP+kDimXP-1, -kDimXYP+kDimXP, -kDimXYP+kDimXP+1,
            /* Group 0, 8 neighbors. */
            -kDimXP-1, -kDimXP, -kDimXP+1,
                   -1,                 +1,
            +kDimXP+1, +kDimXP, +kDimXP+1,
            /* Group +1, 9 neighbors. */
            +kDimXYP-kDimXP-1, +kDimXYP-kDimXP, +kDimXYP-kDimXP+1,
            +kDimXYP       -1, +kDimXYP       , +kDimXYP       +1,
            +kDimXYP+kDimXP-1, +kDimXYP+kDimXP, +kDimXYP+kDimXP+1
    };
    const int frontierLimit = 2;

    for ( int iz = z; iz < blockZEnd; iz += strideZ ) {
        for ( int iy = y; iy < blockYEnd; iy += strideY ) {
            for ( int ix = x; ix < blockXEnd; ix += strideX ) {
                if ( iz < dgd.z && iy < dgd.y && ix < dgd.x )
                {
                    int idx = iz * dimXY + iy * dgd.x + ix;

                    // Load data to the shared memory.
                    shared_mask_array[kIdx] = cmask[idx];

                    if ( threadIdx.x == 0 ) {
                        int kIdxShift = kIdx - 1;
                        if ( ix != 0 ) {
                            shared_mask_array[kIdxShift] = cmask[idx-1];
                        } else {
                            shared_mask_array[kIdxShift] = OCP_MAP_OCC_PADDING;
                        }
                    } else if ( threadIdx.x == blockDim.x - 1 ) {
                        int kIdxShift = kIdx + 1;
                        if ( ix != dgd.x - 1 ) {
                            shared_mask_array[kIdxShift] = cmask[idx+1];
                        } else {
                            shared_mask_array[kIdxShift] = OCP_MAP_OCC_PADDING;
                        }
                    } else if ( ix == dgd.x - 1 ) {
                        shared_mask_array[kIdx + 1] = OCP_MAP_OCC_PADDING;
                    }

                    if ( threadIdx.y == 0 ) {
                        int kIdxShift = kIdx - kDimXP;
                        if ( iy != 0 ) {
                            shared_mask_array[ kIdxShift ] = cmask[ idx - dgd.x ];
                        } else {
                            shared_mask_array[ kIdxShift ] = OCP_MAP_OCC_PADDING;
                        }
                    } else if ( threadIdx.y == blockDim.y - 1 ) {
                        int kIdxShift = kIdx + kDimXP;
                        if ( iy != dgd.y - 1 ) {
                            shared_mask_array[ kIdxShift ] = cmask[ idx + dgd.x ];
                        } else {
                            shared_mask_array[ kIdxShift ] = OCP_MAP_OCC_PADDING;
                        }
                    } else if ( iy == dgd.y - 1 ) {
                        shared_mask_array[ kIdx + kDimXP ] = OCP_MAP_OCC_PADDING;
                    }

                    if ( threadIdx.z == 0 ) {
                        int kIdxShift = kIdx - kDimXYP;
                        if ( iz != 0 ) {
                            shared_mask_array[ kIdxShift ] = cmask[ idx - dimXY ];
                        } else {
                            shared_mask_array[ kIdxShift ] = OCP_MAP_OCC_PADDING;
                        }
                    } else if ( threadIdx.z == blockDim.z - 1 ) {
                        int kIdxShift = kIdx + kDimXYP;
                        if ( iz != dgd.z - 1 ) {
                            shared_mask_array[ kIdxShift ] = cmask[ idx + dimXY ];
                        } else {
                            shared_mask_array[ kIdxShift ] = OCP_MAP_OCC_PADDING;
                        }
                    } else if ( iz == dgd.z - 1 ) {
                        shared_mask_array[ kIdx + kDimXYP ] = OCP_MAP_OCC_PADDING;
                    }

                    __syncthreads();

                    // Check frontier.
                    if ( shared_mask_array[kIdx] == OCP_MAP_OCC_FREE ) {
                        int count = 0;
                        for ( int k = 0; k < nNeighbors; ++k ) {
                            if ( shared_mask_array[ kIdx + neighborShift[k] ] == OCP_MAP_OCC_UNKNOWN ) {
                                count++;
                                if ( count == frontierLimit ) {
                                    cmask[idx] = OCP_MAP_OCC_FRONTIER;
                                    break;
                                }
                            }
                        }
                    }

                    __syncthreads();
                } else {
                    // Load data to the shared memory.
                    shared_mask_array[kIdx] = OCP_MAP_OCC_PADDING;

                    if ( threadIdx.x == 0 ) {
                        int kIdxShift = kIdx - 1;
                        shared_mask_array[kIdxShift] = OCP_MAP_OCC_PADDING;
                    } else if ( threadIdx.x == blockDim.x - 1 ) {
                        int kIdxShift = kIdx + 1;
                        shared_mask_array[kIdxShift] = OCP_MAP_OCC_PADDING;
                    } else if ( ix == dgd.x - 1 ) {
                        shared_mask_array[kIdx + 1] = OCP_MAP_OCC_PADDING;
                    }

                    if ( threadIdx.y == 0 ) {
                        int kIdxShift = kIdx - kDimXP;
                        shared_mask_array[ kIdxShift ] = OCP_MAP_OCC_PADDING;
                    } else if ( threadIdx.y == blockDim.y - 1 ) {
                        int kIdxShift = kIdx + kDimXP;
                        shared_mask_array[ kIdxShift ] = OCP_MAP_OCC_PADDING;
                    } else if ( iy == dgd.y - 1 ) {
                        shared_mask_array[ kIdx + kDimXP ] = OCP_MAP_OCC_PADDING;
                    }

                    if ( threadIdx.z == 0 ) {
                        int kIdxShift = kIdx - kDimXYP;
                        shared_mask_array[ kIdxShift ] = OCP_MAP_OCC_PADDING;
                    } else if ( threadIdx.z == blockDim.z - 1 ) {
                        int kIdxShift = kIdx + kDimXYP;
                        shared_mask_array[ kIdxShift ] = OCP_MAP_OCC_PADDING;
                    } else if ( iz == dgd.z - 1 ) {
                        shared_mask_array[ kIdx + kDimXYP ] = OCP_MAP_OCC_PADDING;
                    }

                    __syncthreads();
                    __syncthreads();
                }
            }
        }
    }
}

void CR_DenseGrid::find_frontiers() {
    // Copy host memory to device.
    thrust::device_vector<CMask> dvDenseGrid = denseGrid;

    DenseGridDim_t dgd;
    dgd.x = nx;
    dgd.y = ny;
    dgd.z = nz;

    // CUDA context check.
    auto err = hipGetLastError();
    if ( hipSuccess != err )
    {
        std::stringstream ss;
        ss << __FILE__ << ": "<< __LINE__ << ": hipGetLastError() returns " << err;
        throw std::runtime_error(ss.str());
    }

    // Launch size.
    const int blockDimX = 8;
    const int blockDimY = 8;
    const int blockDimZ = 8;
    dim3 workingBlockDim(blockDimX, blockDimY, blockDimZ);
    dim3 workingGridDim(8, 8, 8);
    const std::size_t sharedMemSize = ( blockDimZ+2 ) * (blockDimY+2) * (blockDimX+2) * sizeof(CMask);
    g_find_frontiers<<<workingGridDim, workingBlockDim, sharedMemSize>>>(
            thrust::raw_pointer_cast( dvDenseGrid.data() ), dgd );

    // Wait for the GPU.
    hipDeviceSynchronize();

    // CUDA context check.
    err = hipGetLastError();
    if ( hipSuccess != err )
    {
        std::stringstream ss;
        ss << __FILE__ << ": "<< __LINE__ << ": hipGetLastError() returns " << err;
        throw std::runtime_error(ss.str());
    }

    // Copy back from the device.
    denseGrid = dvDenseGrid;
}
